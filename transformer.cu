#include "hip/hip_runtime.h"
#include "transformer.hpp"

void ConstantInit(float *data, int size, float val) {
    for (int i = 0; i < size; ++i) {
        data[i] = val;
    }
}

/**
 * Run a simple test of matrix multiplication using CUDA
 */
int MatrixMultiply(int argc, char **argv,
                   int block_size, const dim3 &dimsA,
                   const dim3 &dimsB) {
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = reinterpret_cast<float *>(malloc(mem_size_A));
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = reinterpret_cast<float *>(malloc(mem_size_B));

    // Initialize host memory
    const float valB = 0.01f;
    ConstantInit(h_A, size_A, 1.0f);
    ConstantInit(h_B, size_B, valB);

    // Allocate device memory
    float *d_A, *d_B, *d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float *h_C = reinterpret_cast<float *>(malloc(mem_size_C));

    if (h_C == NULL) {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));

    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));

    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));

    // Setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

    // Create and start timer
    printf("Computing result using CUDA Kernel...\n");

    // Performs warmup operation using matrixMul CUDA kernel
    if (block_size == 16) {
        MatrixMulCUDA<16> <<< grid, threads >>>(d_C, d_A, d_B,
                                                dimsA.x, dimsB.x);
    } else {
        MatrixMulCUDA<32> <<< grid, threads >>>(d_C, d_A, d_B,
                                                dimsA.x, dimsB.x);
    }

    printf("done\n");

    hipDeviceSynchronize();

    // Allocate CUDA events that we'll use for timing
    hipEvent_t start;
    checkCudaErrors(hipEventCreate(&start));

    hipEvent_t stop;
    checkCudaErrors(hipEventCreate(&stop));

    // Record the start event
    checkCudaErrors(hipEventRecord(start, NULL));

    // Execute the kernel
    int nIter = 300;

    for (int j = 0; j < nIter; j++) {
        if (block_size == 16) {
            MatrixMulCUDA<16> <<< grid, threads >>>(d_C, d_A, d_B,
                                                    dimsA.x, dimsB.x);
        } else {
            MatrixMulCUDA<32> <<< grid, threads >>>(d_C, d_A, d_B,
                                                    dimsA.x, dimsB.x);
        }
    }

    // Record the stop event
    checkCudaErrors(hipEventRecord(stop, NULL));

    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop));

    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    // Compute and print the performance
    float msecPerMatrixMul = msecTotal / nIter;
    double flopsPerMatrixMul = 2.0 * static_cast<double>(dimsA.x) *
                               static_cast<double>(dimsA.y) *
                               static_cast<double>(dimsB.x);
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) /
                       (msecPerMatrixMul / 1000.0f);
    printf(
        "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops," \
        " WorkgroupSize= %u threads/block\n",
        gigaFlops,
        msecPerMatrixMul,
        flopsPerMatrixMul,
        threads.x * threads.y);

    // Copy result from device to host
    checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));

    printf("Checking computed result for correctness: ");
    bool correct = true;

    // test relative error by the formula
    //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
    double eps = 1.e-6;  // machine zero

    for (int i = 0; i < static_cast<int>(dimsC.x * dimsC.y); i++) {
        double abs_err = fabs(h_C[i] - (dimsA.x * valB));
        double dot_length = dimsA.x;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err / abs_val / dot_length;

        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                   i, h_C[i], dimsA.x * valB, eps);
            correct = false;
        }
    }

    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));

    printf("\nNOTE: The CUDA Samples are not meant for performance"\
           "measurements. Results may vary when GPU Boost is enabled.\n");

    if (correct) {
        return EXIT_SUCCESS;
    } else {
        return EXIT_FAILURE;
    }
}


/**
 * Program main
 */
int main(int argc, char **argv) {
    printf("[Linear Layer Using CUDA] - Starting...\n");

    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
            checkCmdLineFlag(argc, (const char **)argv, "?")) {
        printf("Usage -device=n (n >= 0 for deviceID)\n");
        printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
        printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
        printf("  Note: Outer matrix dimensions of A & B matrices" \
               " must be equal.\n");

        exit(EXIT_SUCCESS);
    }

    // This will pick the best possible CUDA capable device, otherwise
    // override the device ID based on input provided at the command line
    int dev = findCudaDevice(argc, (const char **)argv);

    int block_size = 32;

    dim3 dimsA(5 * 2 * block_size, 5 * 2 * block_size, 1);
    dim3 dimsB(5 * 4 * block_size, 5 * 2 * block_size, 1);

    // width of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "wA")) {
        dimsA.x = getCmdLineArgumentInt(argc, (const char **)argv, "wA");
    }

    // height of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "hA")) {
        dimsA.y = getCmdLineArgumentInt(argc, (const char **)argv, "hA");
    }

    // width of Matrix B
    if (checkCmdLineFlag(argc, (const char **)argv, "wB")) {
        dimsB.x = getCmdLineArgumentInt(argc, (const char **)argv, "wB");
    }

    // height of Matrix B
    if (checkCmdLineFlag(argc, (const char **)argv, "hB")) {
        dimsB.y = getCmdLineArgumentInt(argc, (const char **)argv, "hB");
    }

    if (dimsA.x != dimsB.y) {
        printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
               dimsA.x, dimsB.y);
        exit(EXIT_FAILURE);
    }

    printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y,
                                               dimsB.x, dimsB.y);

    int matrix_result = MatrixMultiply(argc, argv, block_size, dimsA, dimsB);

    exit(matrix_result);
}


// void initializeCUDA(int argc, char **argv, int &devID, int &iSizeMultiple, sMatrixSize &matrix_size)
// {
//     // By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
//     hipError_t error;
//     devID = 0;

//     devID = findCudaDevice(argc, (const char **)argv);

//     if (checkCmdLineFlag(argc, (const char **)argv, "sizemult"))
//     {
//         iSizeMultiple = getCmdLineArgumentInt(argc, (const char **)argv, "sizemult");
//     }

//     iSizeMultiple = min(iSizeMultiple, 10);
//     iSizeMultiple = max(iSizeMultiple, 1);

//     hipDeviceProp_t deviceProp;

//     error = hipGetDeviceProperties(&deviceProp, devID);

//     if (error != hipSuccess)
//     {
//         printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
//         exit(EXIT_FAILURE);
//     }

//     printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);

//     int block_size = 32;

//     matrix_size.uiWA = 3 * block_size * iSizeMultiple;
//     matrix_size.uiHA = 4 * block_size * iSizeMultiple;
//     matrix_size.uiWB = 2 * block_size * iSizeMultiple;
//     matrix_size.uiHB = 3 * block_size * iSizeMultiple;
//     matrix_size.uiWC = 2 * block_size * iSizeMultiple;
//     matrix_size.uiHC = 4 * block_size * iSizeMultiple;

//     printf("MatrixA(%u,%u), MatrixB(%u,%u), MatrixC(%u,%u)\n",
//            matrix_size.uiHA, matrix_size.uiWA,
//            matrix_size.uiHB, matrix_size.uiWB,
//            matrix_size.uiHC, matrix_size.uiWC);

//     if( matrix_size.uiWA != matrix_size.uiHB ||
//         matrix_size.uiHA != matrix_size.uiHC ||
//         matrix_size.uiWB != matrix_size.uiWC)
//     {
//        printf("ERROR: Matrix sizes do not match!\n");
//        exit(-1);
//     }
// }

// int main(int argc, char **argv)
// {
//     printf("[Matrix Multiply CUBLAS] - Starting...\n");

//     int devID = 0, sizeMult = 5;
//     sMatrixSize matrix_size;

//     initializeCUDA(argc, argv, devID, sizeMult, matrix_size);

//     // int matrix_result = matrixMultiply(argc, argv, devID, matrix_size);

//     // return matrix_result;
//     return 0;
// }